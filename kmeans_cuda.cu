#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<time.h>
#include<unistd.h>
#include<math.h>
#include<float.h>
#include<errno.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32

int k = 0;
int d = 0;
int MAX_ITER = 0;

void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

const char* getfield(char* line, int num){
    const char* tok;
    for (tok = strtok(line, ",");
            tok && *tok;
            tok = strtok(NULL, ",\n"))
    {
        if (!--num)
            return tok;
    }
    return NULL;
}


float randomInRange(float a, float b){
    return (a + 1) + (((float) rand())/(float) RAND_MAX)*(b-(a+1));
}

void fill(float *array, int size, int val){
    for(int i = 0; i < size; i++){
        array[i] = val;
    }
}

void fillInt(int *array, int size, int val){
    for(int i = 0; i < size; i++){
        array[i] = val;
    }
}

void initialCentres(float *data, float *centres, int size){
    float *minVals = (float*)malloc(d*sizeof(float));
    fill(minVals, d, 100000);
    float *maxVals = (float*)malloc(d*sizeof(float));
    fill(maxVals, d, -100000);
    for(int i = 0; i < size; i++){
        for(int j = 0; j < d; j++){
            if(data[i*d + j] < minVals[j]){
                minVals[j] = data[i*d + j];
            }
            if(data[i*d + j] > maxVals[j]){
                maxVals[j] = data[i*d + j];
            }
        }
    }
    for(int ki = 0; ki < k; ki++){
        for(int i = 0; i < d; i++){
            float r = randomInRange(minVals[i], maxVals[i]);
            centres[ki*d + i] = r;
        }
    }
    free(minVals);
    free(maxVals);
}

float distance(float *a, float *b){
    float dist = 0;
    for(int i = 0; i < d; i++){
        dist += pow(a[i] - b[i], 2);
    }
    return dist;
}

__global__ void assignPoints(float *data, float *centres, float *sum, int *count, int *assignments, int size, int d, int k){
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if(i < size){
        int id = 0;
        float min = 0;
        for(int j = 0; j < d; j++){
            float diff = data[i*d + j] - centres[0 + j];
            min += diff*diff;
        }
        for(int ki = 1; ki < k; ki++){
            float dist = 0;
            for(int j = 0; j < d; j++){
                float diff = data[i*d + j] - centres[ki*d +j];
                dist += diff*diff;
            }
            if(dist < min){
                min = dist;
                id = ki;
            }
        }
        atomicAdd(&count[id], 1);
        assignments[i] = id;
        for(int j = 0; j < d; j++){
            atomicAdd(&sum[id*d + j], data[i*d + j]);
        }
    }
}

__global__ void updateCentres(float *centres, float *sum, int *count, int d){
    int ki = threadIdx.x;
    for(int j = 0; j < d; j++){
        sum[ki*d + j] /= count[ki];
        centres[ki*d + j] = sum[ki*d + j];
    }
}

void print2D(float *data, int size){
    for(int i = 0; i < size; i++){
        for(int j = 0; j < d; j++){
            printf("%f ", data[i*d + j]);
        }
        printf("\n");
    }
}

void print1D(int *data, int size){
    for(int i = 0; i < size; i++){
        printf("%i\n", data[i]);
    }
}
void printMeans(int *data, int size){
    printf("[");
    for(int i = 0; i < size; i++){
        printf("%i", data[i]);
        if(i != size-1){
            printf(", ");
        }
    }
    printf("]\n");
}

void printCount(int *data, int size){
    printf("Count : ");
    for(int i = 0; i < size; i++){
        printf("%i = %i ", i, data[i]);
    }
    printf("\n");
}

int main(int argc, char *argv[]){
    if(argc != 5){
        if(argc < 5){
            printf("Not enough arguments given.\n");
        } else{
            printf("Too many arguments given.\n");
        }
        printf("Please input the following arguments in the given order:\n");
        printf("  -<file_name>.csv\n");
        printf("  -the number of rows to read from csv\n");
        printf("  -the number of clusters to create\n");
        printf("  -the number of iterations to run\n");
        exit(-1);
    } else if(strstr(argv[1], ".csv") == NULL){
        printf("File name entered is not a csv.\n");
        exit(-1);
    }
    
    d = atoi(argv[2]);
    k = atoi(argv[3]);
    MAX_ITER = atoi(argv[4]);
    
    srand((unsigned int)time(NULL));
    double time = 0.0;
    clock_t start, end;
    FILE* cf = fopen(argv[1], "r");
    FILE* fp = fopen(argv[1], "r");
    hipEvent_t launch_begin_seq, launch_end_seq;
    if(!fp){
        perror("fopen");
        exit(-1);
    }
    if(!cf){
        perror("fopen");
        exit(-1);
    }
    char line[1024];
    int size = 0;
    while(fgets(line, 1024, cf)){
        size++;
    }
    
    float *data = (float*)malloc(size*d*sizeof(float));
    int j = 0;
    while(fgets(line, 1024, fp)){
        for(int i = 1; i < d + 1; i++){
            char* tmp = strdup(line);
            data[j*d + (i - 1)] = atof(getfield(tmp, i));
            free(tmp);
        }
        j++;
    }
    
    fclose(cf);
    fclose(fp);
    
    float *centres = (float*)malloc(k*d*sizeof(float));
    int *count = (int*)malloc(k*sizeof(int));
    int *assignments = (int*)malloc(size*sizeof(int));
    
    float *d_data, *d_centres, *d_sum;
    int *d_count, *d_assignments;
    hipMalloc((void**)&d_data, size*d*sizeof(float));
    hipMalloc((void**)&d_centres, k*d*sizeof(float));
    hipMalloc((void**)&d_sum, k*d*sizeof(float));
    hipMalloc((void**)&d_count, k*sizeof(int));
    hipMalloc((void**)&d_assignments, size*sizeof(int));
    
    hipMemcpy(d_data, data, size*d*sizeof(float), hipMemcpyHostToDevice);
    
    int ts = size / BLOCK_SIZE;
    if(size % BLOCK_SIZE != 0) ts++;

    dim3 dimGrid(ts);
    dim3 dimBlock(BLOCK_SIZE);
    
    initialCentres(data, centres, size);
    hipMemcpy(d_centres, centres, k*d*sizeof(float), hipMemcpyHostToDevice);
    
    start = clock();
    
    int iter = 0;
    while(iter < MAX_ITER){
        hipEventCreate(&launch_begin_seq);
        hipEventCreate(&launch_end_seq);
        hipMemset(d_count, 0, k*sizeof(int));
        hipMemset(d_sum, 0, k*d*sizeof(float));
        
        hipEventRecord(launch_begin_seq,0);
        assignPoints<<<dimGrid, dimBlock>>>(d_data, d_centres, d_sum, d_count, d_assignments, size, d, k);
        hipEventRecord(launch_end_seq,0);

        hipEventSynchronize(launch_end_seq);
        
        checkCUDAError("assignPoints");
        
        hipEventCreate(&launch_begin_seq);
        hipEventCreate(&launch_end_seq);

        hipEventRecord(launch_begin_seq,0);
        updateCentres<<<1, k>>>(d_centres, d_sum, d_count, d);
        hipEventRecord(launch_end_seq,0);

        hipEventSynchronize(launch_end_seq);
        
        checkCUDAError("updateCentres");
        
        iter++;
    }
    end = clock();
    
    time += (double) (end - start) / CLOCKS_PER_SEC;
    
    printf("Time = %f seconds\n", time);
    
    hipMemcpy(centres, d_centres, k*d*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(count, d_count, k*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(assignments, d_assignments, size*sizeof(int), hipMemcpyDeviceToHost);
    printf("Centres:\n");
    print2D(centres, k);
    printCount(count, k);
    //printMeans(assignments, size);
    
    free(data);
    free(centres);
    free(count);
    free(assignments);
    hipFree(d_data);
    hipFree(d_centres);
    hipFree(d_sum);
    hipFree(d_count);
    hipFree(d_assignments);
}
